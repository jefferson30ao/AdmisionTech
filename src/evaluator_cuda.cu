#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> // ceil
#include <cstdint>
#include <iostream>

#include "../backend/include/evaluator.hpp"

// Definición de la clave en memoria constante
__constant__ int8_t d_key[100];

extern "C" void set_key_cuda(const int8_t* h_key) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_key), h_key, 100 * sizeof(int8_t), 0, hipMemcpyHostToDevice);
}

namespace exam {

// Kernel CUDA para evaluar las respuestas de los estudiantes
__global__ void evaluateKernel(const int8_t* d_answers, size_t num_students, size_t num_questions, ScoringRule rule, Result* d_results) {
    size_t student_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (student_idx < num_students) {
        uint32_t correct_count = 0;
        uint32_t wrong_count = 0;
        uint32_t blank_count = 0;
        double score = 0.0;

        for (size_t i = 0; i < num_questions; ++i) {
            int8_t student_answer = d_answers[student_idx * num_questions + i];
            int8_t correct_answer = d_key[i];

            if (student_answer == correct_answer) {
                correct_count++;
                score += rule.correct;
            } else if (student_answer == -1) {
                blank_count++;
                score += rule.blank;
            } else {
                wrong_count++;
                score += rule.wrong;
            }
        }

        d_results[student_idx].score = score;
        d_results[student_idx].correct = correct_count;
        d_results[student_idx].wrong = wrong_count;
        d_results[student_idx].blank = blank_count;
    }
}

// Función pública para invocar el kernel CUDA
void evaluate_cuda(const int8_t* h_answers, size_t num_students, const int8_t* h_key, size_t num_questions, ScoringRule rule, Result* h_results) {
    std::cout << "[CUDA] Evaluating " << num_students << " students with " << num_questions << " questions\n";
    set_key_cuda(h_key);

    int8_t* d_answers = nullptr;
    Result* d_results = nullptr;

    size_t answers_bytes = num_students * num_questions * sizeof(int8_t);
    size_t results_bytes = num_students * sizeof(Result);

    hipMalloc((void**)&d_answers, answers_bytes);
    hipMalloc((void**)&d_results, results_bytes);

    hipMemcpy(d_answers, h_answers, answers_bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (int)ceil((float)num_students / threadsPerBlock);

    evaluateKernel<<<blocksPerGrid, threadsPerBlock>>>(d_answers, num_students, num_questions, rule, d_results);
    hipDeviceSynchronize();

    hipMemcpy(h_results, d_results, results_bytes, hipMemcpyDeviceToHost);

    hipFree(d_answers);
    hipFree(d_results);

    std::cout << "[CUDA] Evaluation complete\n";
}

} // namespace exam